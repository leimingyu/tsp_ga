#include "hip/hip_runtime.h"
#include <iostream>
#include <algorithm>
#include <functional> // for bind()

// Program includes
#include "ga_gpu.h"
#include "common.h"

using namespace std;

bool checkForKernelError(const char *err_msg)
{
		hipError_t status = hipGetLastError();
		if (status != hipSuccess)
		{
				cout << err_msg << hipGetErrorString(status) << endl;
				return true;
		}
		else
		{
				return false;
		}
}

__device__ int getGlobalIdx_2D_1D()
{
		int blockId  = blockIdx.y * gridDim.x + blockIdx.x;			 	
		int threadId = blockId * blockDim.x + threadIdx.x; 
		return threadId;
}

__device__ void crossover(World* old_pop, World* new_pop, int* sel_ix,  int* cross_loc, int tid)
{

		// Copy elements from first parent up through crossover point
		memcpy(new_pop[tid].cities, old_pop[sel_ix[2*tid]].cities,(cross_loc[tid] + 1) * sizeof(City));

		// Add remaining elements from second parent to child, in order
		int remaining = old_pop[tid].num_cities - cross_loc[tid] - 1;
		int count     = 0;
		for (int i=0; i<old_pop[tid].num_cities; i++)
		{
				bool in_child = false;
				for (int j=0; j<=cross_loc[tid]; j++)     
				{
						// If the city is in the child, exit
						if ((new_pop[tid].cities[j].x == old_pop[sel_ix[2 * tid + 1]].cities[i].x) &               
										(new_pop[tid].cities[j].y == old_pop[sel_ix[2 * tid + 1]].cities[i].y))
						{
								in_child = true;
								break;
						}
				}
				if (!in_child)
				{
						count++;
						memcpy(&new_pop[tid].cities[cross_loc[tid] + count],&old_pop[sel_ix[2 * tid + 1]].cities[i], sizeof(City));
				}

				// Stop once all of the cities have been added
				if (count == remaining) break;
		}
}

__device__ void mutate(World* new_pop, int* mutate_loc, int tid)
{
		// Swap the elements
		City temp = *(new_pop[tid].cities + mutate_loc[2*tid]);
		*(new_pop[tid].cities + mutate_loc[2*tid])   = *(new_pop[tid].cities + mutate_loc[2*tid+1]);
		*(new_pop[tid].cities + mutate_loc[2*tid+1]) = temp;
}

__global__ void fitness_kernel(World* pop, int pop_size)
{
		// Get the thread id
		//int tid = getGlobalIdx_2D_1D();
		int tid = threadIdx.x + blockIdx.x * blockDim.x;

		// Evaluate if the thread is valid
		if (tid < pop_size)
		{
				float distance = 0.f; // Total "normalized" "distance"

				// Calculate fitnesses using the fitness formula that will be explained in the reports
				int city_num = pop[tid].num_cities - 1;
				for (int i=0; i<city_num; i++) {
						float loc_x = pop[tid].cities[i].x - pop[tid].cities[i + 1].x;
						float loc_y = pop[tid].cities[i].y - pop[tid].cities[i + 1].y;
						distance += loc_x * loc_x + loc_y * loc_y;
				}

				pop[tid].fitness = (pop[tid].width * pop[tid].height) / distance;
		}
}

//
// NOTE: histogram for better performance 
//
__global__ void fit_sum_kernel(World* pop, int pop_size, float* fit_sum)
{
		int tid = getGlobalIdx_2D_1D();

		if (tid < pop_size)
		{
				// Sum of all fitness
				float sum = 0.f;

				// Calculate the partial sum
				for (int i=0; i<=tid; i++)
						sum += pop[i].fitness;

				pop[tid].fit_prob = sum;

				if (tid == (pop_size - 1))	*fit_sum = sum;
		}
}


//
// NOTE: use cooperative groups to merge two kernels togather :  fit_sum_kernel + fit_prob_kernel 
//

__global__ void fit_prob_kernel(World* pop, int pop_size, float* fit_sum)
{
		// Get the thread id
		int tid = getGlobalIdx_2D_1D();

		// Evaluate if the thread is valid
		if (tid < pop_size)
				pop[tid].fit_prob /= *fit_sum;
}

__global__ void max_fit_kernel(World* pop, int pop_size, World* gen_leader)
{
		// Get the thread id
		int tid = getGlobalIdx_2D_1D();

		// Evaluate if the thread is valid
		if (tid < pop_size)
		{
				if (tid == 0)
				{
						float max = (float)0.0;
						int ix  = 0;
						for (int i=1; i<pop_size; i++)
						{
								if (pop[i].fitness > max)
								{
										max = pop[i].fitness;
										ix  = i;
								}
						}
						gen_leader->cities  = pop[ix].cities;
						gen_leader->fitness = max;
				}
				else if (tid == 1)
				{
						gen_leader->height     = pop[0].height;
						gen_leader->width      = pop[0].width;
						gen_leader->num_cities = pop[0].num_cities;
				}
		}
}

__global__ void selection_kernel(World* pop, int pop_size, float* rand_nums,  \
				int* sel_ix)
{
		// Get the thread id
		int tid = getGlobalIdx_2D_1D();

		// Evaluate if the thread is valid
		if (tid < (2 * pop_size))
		{
				// Select the parents
				for (int j=0; j<pop_size; j++)
				{
						if (rand_nums[tid] <= pop[j].fit_prob)
						{
								sel_ix[tid] = j;
								break;
						}
				}
		}
}

__global__ void child_kernel(World* old_pop, World* new_pop, int pop_size,    \
				int* sel_ix, float prob_crossover, float* prob_cross, int* cross_loc,     \
				float prob_mutation, float* prob_mutate, int* mutate_loc)
{
		// Get the thread id
		int tid = getGlobalIdx_2D_1D();

		// Evaluate if the thread is valid
		if (tid < pop_size)
		{
				// Determine how many children are born
				if (prob_cross[tid] <= prob_crossover)
				{
						crossover(old_pop, new_pop, sel_ix, cross_loc, tid);
						if(prob_mutate[tid] <= prob_mutation)
								mutate(new_pop, mutate_loc, tid);

				}
				else // Select the first parent
				{
						// Add child to new population
						memcpy(new_pop[tid].cities, old_pop[sel_ix[2*tid]].cities, old_pop[tid].num_cities * sizeof(City));
						if(prob_mutate[tid] <= prob_mutation)
								mutate(new_pop, mutate_loc, tid);
				}
		}
}

bool g_initialize(World* world, World* pop_d, int pop_size, int seed)
{
		bool error;
		World h_world;
		h_world.cities = new City[world->num_cities * sizeof(City)];
		srand(seed);
		for (int i=0; i<pop_size; i++)
		{
				clone_world(world, &h_world);
				// Randomly adjust the path between cities
				random_shuffle(&h_world.cities[0], &h_world.cities[world->num_cities]);

				// Copy world to device
				error = g_soft_clone_world(&pop_d[i], &h_world);
				if (error)
						return true;

				// Allocate space for cities on device
				City *d_city;
				error = checkForError(hipMalloc((void**)&d_city, world->num_cities * sizeof(City)));
				if (error)
						return true;

				// Copy cities to device
				error = checkForError(hipMemcpy(d_city, h_world.cities,world->num_cities * sizeof(City), hipMemcpyHostToDevice));
				if (error)
						return true;

				// Update pointer on device
				error = checkForError(hipMemcpy(&pop_d[i].cities, &d_city, sizeof(City*), hipMemcpyHostToDevice));
				if (error)
						return true;
		}

		// Success
		delete[] h_world.cities; return false;
}

bool g_evaluate(World *pop, int pop_size, dim3 Block, dim3 Grid, int blk_size, int grid_size)
{
		bool error;

		// Allocate fitness sum on the GPU
		float *fit_sum_d;
		error = checkForError(hipMalloc((void**)&fit_sum_d, sizeof(float)));
		if (error)
				return true;

		// Calculate the fitnesses
		//fitness_kernel <<< Grid, Block >>> (pop, pop_size);
		fitness_kernel <<< grid_size, blk_size >>> (pop, pop_size);
		//hipDeviceSynchronize();

		if (checkForKernelError("fitness_kernel is failing "))
				return true;

		// Calculate the total sum and compute the partial probabilities
		fit_sum_kernel <<< Grid, Block >>> (pop, pop_size, fit_sum_d);
		//hipDeviceSynchronize();

		if (checkForKernelError("fit_sum_kernel failing "))
				return true;

		// Compute the full probabilities
		fit_prob_kernel <<< Grid, Block >>> (pop, pop_size, fit_sum_d);
		//hipDeviceSynchronize();
		if (checkForKernelError("fit_prob_kernel failing "))
				return true;

		hipFree(fit_sum_d); 
		return false;
}

int g_select_leader(World* pop, int pop_size, World* generation_leader,
				World* best_leader, dim3 Block, dim3 Grid)
{
		// Error handling
		bool error;

		// Initialize world for device generation leader
		World *gen_leader_d;
		error = checkForError(hipMalloc((void**)&gen_leader_d, sizeof(World)));
		if (error)
				return -1;
		// Purposely don't allocate space for the cities, simply use a pointer.
		// Since we only need to copy data to cpu 
		error = g_soft_clone_world(gen_leader_d, generation_leader);
		if (error)
				return -1;

		// Calculate the max fitness
		max_fit_kernel <<< Grid, Block >>> (pop, pop_size, gen_leader_d);
		hipDeviceSynchronize();
		if (checkForKernelError("*** Max fitness kernel failed: "))
				return -1;

		// Copy results from device
		City *h_ptr = generation_leader->cities;
		City *d_ptr;
		error = checkForError(hipMemcpy(generation_leader, gen_leader_d, sizeof(World), hipMemcpyDeviceToHost));
		if (error)
				return -1;
		d_ptr = generation_leader->cities;
		generation_leader->cities = h_ptr;
		error = checkForError(hipMemcpy(generation_leader->cities,  d_ptr, generation_leader->num_cities * sizeof(City), hipMemcpyDeviceToHost));
		if (error)
				return -1;

		// Update best leader
		if (generation_leader->fitness > best_leader->fitness)
		{
				clone_world(generation_leader, best_leader);
				hipFree(gen_leader_d); return 1;
		}

		// Success
		hipFree(gen_leader_d);	return 0;
}

bool g_execute(float prob_mutation, float prob_crossover, int pop_size,
				int max_gen, World* world, int seed, int tile_size, int grid_size, int grid_size2,
				int pop_bytes)
{
		// Error checking variables
		bool error;

		// Random number generation
		mt19937::result_type rseed = seed;
		auto rgen = bind(uniform_real_distribution<>(0, 1), mt19937(rseed));

		dim3 Block(tile_size);
		dim3 Grid(grid_size, grid_size);
		dim3 Grid2(grid_size2, grid_size2);

		World *old_pop_d, *new_pop_d;

		// Random numbers
		float *prob_select = new float[2 * pop_size * sizeof(float)];
		float *prob_cross  = new float[pop_size * sizeof(float)];
		float *prob_mutate = new float[pop_size * sizeof(float)];
		int   *cross_loc   = new int[pop_size * sizeof(int)];
		int   *mutate_loc  = new int[2 * pop_size * sizeof(int)];
		float *prob_select_d, *prob_cross_d, *prob_mutate_d;
		int   *cross_loc_d, *mutate_loc_d;

		int   sel;
		int   best_generation    = 0;
		World *best_leader       = new World[sizeof(World)];
		World *generation_leader = new World[sizeof(World)];

		// Other "temporary" parameters
		int *sel_ix_d;

		//Initializations
		init_world(best_leader, world->width, world->height, world->num_cities);
		init_world(generation_leader, world->width, world->height, world->num_cities);

		if (checkForError(hipMalloc((void**) &old_pop_d, pop_bytes)))
				return true;
		if (checkForError(hipMalloc((void**) &new_pop_d, pop_bytes)))
				return true;
		if (checkForError(hipMalloc((void**) &prob_select_d, sizeof(float)* 2 * pop_size)))
				return true;
		if (checkForError(hipMalloc((void**) &prob_cross_d, sizeof(float) * pop_size)))
				return true;
		if (checkForError(hipMalloc((void**) &prob_mutate_d, sizeof(float) * pop_size)))
				return true;
		if (checkForError(hipMalloc((void**) &cross_loc_d, sizeof(int) * pop_size)))
				return true;
		if (checkForError(hipMalloc((void**) &mutate_loc_d, sizeof(int) * 2 * pop_size)))
				return true;
		// Other parameters
		if (checkForError(hipMalloc((void**) &sel_ix_d, sizeof(int) * 2 * pop_size)))
				return true;

		///////// GPU Initializations
		// Populations
		error = g_initialize(world, old_pop_d, pop_size, seed);
		if (error)
				return true;
		for (int i=0; i<pop_size; i++)
		{
				error = g_init_world(&new_pop_d[i], world);
				if (error)
						return true;
		}

		// Calculate the fitnesses
		error = g_evaluate(old_pop_d, pop_size, Block, Grid, tile_size, grid_size);
		if (error)
				return true;

		// Initialize the best leader
		sel = g_select_leader(old_pop_d, pop_size, generation_leader,             \
						best_leader, Block, Grid);	
		if (-1 == sel)
				return true;	


		print_status(generation_leader, best_leader, 0);

		// Continue through all generations
		for (int i=0; i<max_gen; i++)
		{

				// Generate all probabilities for each step
				// Note : The order the random numbers are generated must be consistent to
				// ensure the results will match the CPU.
				for (int j=0; j<pop_size; j++)
				{
						prob_select[2*j]     = (float)rgen();
						prob_select[2*j + 1] = (float)rgen();
						prob_cross[j]        = (float)rgen();
						cross_loc[j]         = (int)(rgen() * (world->num_cities - 1));
						prob_mutate[j]       = (float)rgen();
						mutate_loc[2*j]      = (int)(rgen() * (world->num_cities));
						mutate_loc[2*j + 1]  = (int)(rgen() * (world->num_cities));
						while (mutate_loc[2*j + 1] == mutate_loc[2*j])
						{
								mutate_loc[2*j + 1] = (int)(rgen() * world->num_cities);
						}
				}

				// Copy random numbers to device
				if (checkForError(hipMemcpy(prob_select_d, prob_select, 2 * pop_size * sizeof(float), hipMemcpyHostToDevice)))
						return true;
				if (checkForError(hipMemcpy(prob_cross_d, prob_cross, pop_size * sizeof(float), hipMemcpyHostToDevice)))
						return true;
				if (checkForError(hipMemcpy(prob_mutate_d, prob_mutate,pop_size * sizeof(float), hipMemcpyHostToDevice)))
						return true;
				if (checkForError(hipMemcpy(cross_loc_d, cross_loc,pop_size * sizeof(int), hipMemcpyHostToDevice)))
						return true;
				if (checkForError(hipMemcpy(mutate_loc_d, mutate_loc, 2 * pop_size * sizeof(int), hipMemcpyHostToDevice)))
						return true;

				// Select the parents
				selection_kernel <<< Grid2, Block >>> (old_pop_d, pop_size, prob_select_d, sel_ix_d);
				hipDeviceSynchronize();
				if (checkForKernelError("slection_kernel is failing "))
						return true;

				// Create the children (form the new population entirely on the GPU!)
				child_kernel <<< Grid, Block >>> (old_pop_d, new_pop_d, pop_size, sel_ix_d, prob_crossover, prob_cross_d, cross_loc_d, prob_mutation, prob_mutate_d, mutate_loc_d);
				hipDeviceSynchronize();
				if (checkForKernelError("child_kernel is failing"))
						return true;

				// Calculate the fitnesses on the new population
				error = g_evaluate(new_pop_d, pop_size, Block, Grid, tile_size, grid_size);
				if (error)
						return true;	

				// Swap the populations
				World* temp_d = old_pop_d;
				old_pop_d     = new_pop_d;
				new_pop_d     = temp_d;

				// Select the new leaders
				sel = g_select_leader(old_pop_d, pop_size, generation_leader,best_leader, Block, Grid);
				if (-1 == sel)
						return true;
				else if (1 == sel) best_generation = i + 1;
				print_status(generation_leader, best_leader, i + 1);
		} 

		cout << endl << "Best generation found at " << best_generation << " generations" << endl;

		//freeing memory
		delete[] prob_select; delete[] prob_cross; delete[] prob_mutate;
		delete[] cross_loc; delete[] mutate_loc; free_world(best_leader);
		free_world(generation_leader); hipFree(old_pop_d); hipFree(cross_loc_d); 
		hipFree(new_pop_d); hipFree(prob_select_d); hipFree(prob_cross_d);
		hipFree(prob_mutate_d); hipFree(mutate_loc_d); hipFree(sel_ix_d);	
		return false;
}
